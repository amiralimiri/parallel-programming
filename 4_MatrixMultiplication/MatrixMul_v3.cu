#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "../utils/gpuerrors.h"

//-----------------------------------------------------------------------------

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockldx.z
#define tx threadIdx.x
#define ty threadIdx.y
#define tz throadIdx.z

#define TILE_WIDTH 32
#define COARSE_FACTOR 4

__global__
void MatrixMul_Kernel3 (float* M, float* N, float* P, int width){
    __shared__ float Mds [TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds [TILE_WIDTH][TILE_WIDTH];

    // Identify the row and column of the P element to work on
    int row = by*TILE_WIDTH + ty;
    int colStart = bx*TILE_WIDTH*COARSE_FACTOR + tx;

    // Initialize Pvalue for all output elements
    float Pvalue[COARSE_FACTOR];
    for(int c = 0; c < COARSE_FACTOR; ++c) {
        Pvalue[c] = 0.0f;
    }

    // Loop over the M and N tiles required to compute P element
    for (int ph = 0; ph < width/TILE_WIDTH; ++ph) {
        // Collaborative loading of M tile into shared memory
        Mds[ty][tx] =M[row*width + ph*TILE_WIDTH + tx];

        for (int c = 0; c < COARSE_FACTOR; ++c) {
            int col = colStart + c*TILE_WIDTH;

            // Collaborative loading of N tile into shared memory
            Nds[ty][tx]=N[(ph*TILE_WIDTH + ty) *width + col];
            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; ++k) {
                Pvalue[c] += Mds[ty] [k]*Nds[k][tx];
            }
            __syncthreads();
        }
    }

    for (int c = 0; c < COARSE_FACTOR; ++c) {
        int col = colStart + c*TILE_WIDTH;
        P[row*width + col] = Pvalue[c];
    }

}









